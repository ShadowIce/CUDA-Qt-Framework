#include <hip/hip_runtime.h>

// Kernel definition
__global__ void VecAdd(int n, float* v1, float* v2, float* v3) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n) v3[i] = v1[i] + v2[i];
}

// exported function, usable in the C++ part
extern "C" void RunVecAdd(int n, float* v1, float* v2, float* v3) {
	size_t size = n * sizeof(float);
	
	float* d_v1;
	float* d_v2;
	float* d_v3;

	hipMalloc((void**)&d_v1, size);
	hipMalloc((void**)&d_v2, size);
	hipMalloc((void**)&d_v3, size);

	hipMemcpy(d_v1, v1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_v2, v2, size, hipMemcpyHostToDevice);

	int threadsPerBlock = 256;
	int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
	
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(n, d_v1, d_v2, d_v3);

	hipMemcpy(v3, d_v3, size, hipMemcpyDeviceToHost);
	
	hipFree(d_v1);
	hipFree(d_v2);
	hipFree(d_v3);
}
